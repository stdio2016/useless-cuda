
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<cstdint>

typedef uint32_t u32;

__global__ void gen(u32 *src, int nsrc, u32 *choice, int nchoices, u32 *dest, int bufsize, int *ngen) {
  __shared__ u32 some[256];
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int nthreads = blockDim.x * gridDim.x;
  for (int i = threadIdx.x; i < nchoices; i += blockDim.x) {
    some[i] = choice[i];
  }
  __syncthreads();
  int sum = 0;
  for (int i = tid; i < nsrc; i += nthreads) {
    u32 a = src[i];
    for (int j = 0; j < nchoices; j++) {
      if ((a & some[j]) == 0) {
        int at = atomicAdd(ngen, 1);
        if (at >= bufsize) goto bye;
        dest[at] = a + some[j];
      }
    }
  }
  dest[tid] = sum;
  return;
  bye:
  dest[tid] = 0;
}


int main() {
  int nsrc = 1000000, nchoices = 100, bufsize = 30000000;
  u32 *gsrc, *gchoice, *gdest;
  int *gcnt;
  hipMalloc(&gsrc, sizeof(u32) * nsrc);
  hipMalloc(&gchoice, sizeof(u32) * nchoices);
  hipMalloc(&gdest, sizeof(u32) * bufsize);
  hipMalloc(&gcnt, sizeof(int));
  u32 *src, *choice, *dest;
  src = new u32[nsrc];
  choice = new u32[nchoices];
  dest = new u32[bufsize];
  unsigned seed = 2;
  for (int i = 0; i < nsrc; i++) {
    src[i] = seed;
    seed = seed*0xdefaced + 1;
  }
  for (int i = 0; i < nchoices; i++) {
    u32 rr = seed;
    seed = seed*0xdefaced + 1;
    choice[i] = 0;
    for (int yy = 0; yy < 5; yy++) {
      choice[i] += 1<<(rr>>yy*5&31);
    }
  }
  hipMemcpy(gsrc, src, sizeof(u32) * nsrc, hipMemcpyHostToDevice);
  hipMemcpy(gchoice, choice, sizeof(u32) * nchoices, hipMemcpyHostToDevice);
  hipMemset(gcnt, 0, sizeof(int));
  int ans = 0;
  gen<<<40, 256>>>(gsrc, nsrc, gchoice, nchoices, gdest, bufsize, gcnt);
  hipMemcpy(&ans, gcnt, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(dest, gdest, sizeof(u32) * ans, hipMemcpyDeviceToHost);
  printf("generated: %d\n", ans);
}
