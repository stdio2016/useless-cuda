
#include <hip/hip_runtime.h>
#include <stdio.h>
#include<chrono>
int main() {
    auto t0 = std::chrono::steady_clock::now();
    hipEvent_t a;
    hipEventCreate(&a);
    auto t1 = std::chrono::steady_clock::now();
    std::chrono::duration<double> diff = t1 - t0;
    printf("init takes %fs\n", diff.count());
}
