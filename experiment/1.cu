// get cuda max hardware concurrency
// by stdio2016 2023-03-18
#include<hip/hip_runtime.h>
#include<stdio.h>
__device__ int current_concurrency = 0;
__device__ void waitClockGpu(int time) {
    long long t0 = clock64();
    while (clock64() - t0 < time) {
        ;
    }
}
__global__ void concurrency_test(__global__ int *max_concurrency) {
    atomicAdd(&current_concurrency, 1);
    waitClockGpu(1000000);
    atomicMax(max_concurrency, current_concurrency);
    waitClockGpu(1000000);
    atomicAdd(&current_concurrency, -1);
}
int main(int a, char*b[]){
    int *max_concurrency;
    hipMalloc(&max_concurrency, sizeof(int));
    int num_streams = 0;
    printf("number of streams: ");
    scanf("%d", &num_streams);
    hipStream_t *ts = new hipStream_t[num_streams];
    for (int i = 0; i < num_streams; i++) {
        hipStreamCreate(&ts[i]);
    }
    for (int i = 0; i < num_streams; i++) {
        for (int j = 0; j < 100; j++) {
            concurrency_test<<<100, 256, 0, ts[i]>>>(max_concurrency);
        }
        //cudaStreamQuery(ts[i]);
        printf("stream %d sent\n", i);
    }
    for (int i = 0; i < num_streams; i++) {
        hipStreamSynchronize(ts[i]);
        printf("stream %d synchronized\n", i);
    }
    int num = 0;
    hipMemcpy(&num, max_concurrency, sizeof(int), hipMemcpyDeviceToHost);
    printf("max concurrency: %d\n", num);
    
    for (int i = 0; i < num_streams; i++) {
        hipStreamDestroy(ts[i]);
    }
    delete[] ts;
    hipFree(max_concurrency);
}
