
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    long long kb = 0;
    printf("how many kb of pinned memory: ");
    scanf("%lld", &kb);
    void *dat;
    hipError_t status = hipHostMalloc(&dat, (size_t)kb * 1024, hipHostMallocPortable, hipHostMallocDefault);
    printf("status: %d\n", status);
    hipHostFree(dat);
}
