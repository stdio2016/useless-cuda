// test cuda launch speed in device

#include <hip/hip_runtime.h>
#include<stdio.h>
#include<cstring>

__global__ void emptyKernel() {
}

__global__ void cdpKernel() {
    emptyKernel<<<1,1>>>();
    emptyKernel<<<1,1>>>();
    emptyKernel<<<1,1>>>();
    emptyKernel<<<1,1>>>();
    emptyKernel<<<1,1>>>();
    emptyKernel<<<1,1>>>();
    emptyKernel<<<1,1>>>();
    emptyKernel<<<1,1>>>();
    emptyKernel<<<1,1>>>();
}

int main(int argc, char *argv[]) {
    bool sync = false;
    if (argc >= 2) {
        if (strcmp(argv[1], "sync") == 0) {
            sync = true;
        }
    }
    if (sync) {
        puts("use hipDeviceSynchronize");
    }
    else {
        puts("no hipDeviceSynchronize");
    }
    for (int j=0;j<10;j++){
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);
    hipEventRecord(event1, 0);
    for (int i = 0; i < 10000; i++) {
        cdpKernel<<<1,1>>>();
        if (sync) hipDeviceSynchronize();
    }
    hipDeviceSynchronize();
    hipEventRecord(event2, 0);
    hipEventSynchronize(event2);
    float timeMs;
    hipEventElapsedTime(&timeMs, event1, event2);
    printf("time %fms (%f launches per sec)\n", timeMs, 1e5*1000.0/timeMs);
    hipEventDestroy(event1);
    hipEventDestroy(event2);
    }
}
